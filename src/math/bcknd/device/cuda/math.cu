
#include <hip/hip_runtime.h>
/**
 * Device kernel for add2s1
 */
__global__ void add2s1_kernel(double * __restrict__ a,
			      const double * __restrict__ b,
			      const double c1,
			      const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += str) {
    a[i] = c1 * a[i] + b[i];
  }
}

/**
 * Device kernel for add2s2
 */
__global__ void add2s2_kernel(double * __restrict__ a,
			      const double * __restrict__ b,
			      const double c1,
			      const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += str) {
    a[i] = a[i] + c1 * b[i];
  }
}

/**
 * Device kernel for invcol2
 */
__global__ void invcol2_kernel(double * __restrict__ a,
			       const double * __restrict__ b,
			       const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;
  
  for (int i = idx; i < n; i += str) {
    a[i] = a[i] / b[i];
  }  
}

/** 
 * Device kernel for col2
 */
__global__ void col2_kernel(double * __restrict__ a,
			    const double * __restrict__ b,
			    const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += str) {
    a[i] = a[i] * b[i];
  }  
}

/** 
 * Device kernel for col3
 */
__global__ void col3_kernel(double * __restrict__ a,
			    const double * __restrict__ b,
			    const double * __restrict__ c,
			    const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += str) {
    a[i] = b[i] * c[i];
  }  
}

/** 
 * Device kernel for sub3
 */
__global__ void sub3_kernel(double * __restrict__ a,
			    const double * __restrict__ b,
			    const double * __restrict__ c,
			    const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += str) {
    a[i] = b[i] - c[i];
  }  
}

/**
 * Device kernel for addcol3
 */
__global__ void addcol3_kernel(double * __restrict__ a,
			    const double * __restrict__ b,
			    const double * __restrict__ c,
			    const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += str) {
    a[i] = a[i] + b[i] * c[i];
  }  
  

}

/**
 * Device kernel for glsc3
 */
__global__ void glsc3_kernel(const double * a,
			     const double * b,
			     const double * c,
			     double * buf_h,
			     const int n) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  __shared__ double buf[1024];
  double tmp = 0.0;

  for (int i = idx; i < n; i+= str) {
    tmp += a[i] * b[i] * c[i];
  }
  buf[threadIdx.x] = tmp;
  __syncthreads();

  int i = blockDim.x>>1;
  while (i != 0) {
    if (threadIdx.x < i) {
      buf[threadIdx.x] += buf[threadIdx.x + i];
    }
    __syncthreads();
    i = i>>1;
  }
 
  if (threadIdx.x == 0) {
    buf_h[blockIdx.x] = buf[0];
  }
}

extern "C" {

  /** Fortran wrapper for copy
   * Copy a vector \f$ a = b \f$
   */
  void cuda_copy(void *a, void *b, int *n) {
    hipMemcpy(a, b, (*n) * sizeof(double), hipMemcpyDeviceToDevice);
  }

  /** Fortran wrapper for rzero
   * Zero a real vector
   */
  void cuda_rzero(void *a, int *n) {
    hipMemset(a, 0, (*n) * sizeof(double));
  }

  
  /**
   * Fortran wrapper for add2s1
   * Vector addition with scalar multiplication \f$ a = c_1 a + b \f$
   * (multiplication on first argument) 
   */
  void cuda_add2s1(void *a, void *b, double *c1, int *n) {
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    add2s1_kernel<<<nblcks, nthrds>>>((double *) a,
				      (double *) b,
				      *c1, *n);
    
  }

  /**
   * Fortran wrapper for add2s2
   * Vector addition with scalar multiplication \f$ a = a + c_1 b \f$
   * (multiplication on second argument) 
   */
  void cuda_add2s2(void *a, void *b, double *c1, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    add2s2_kernel<<<nblcks, nthrds>>>((double *) a,
				      (double *) b,
				      *c1, *n);

  }

  /**
   * Fortran wrapper for invcol2
   * Vector division \f$ a = a / b \f$
   */
  void cuda_invcol2(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    invcol2_kernel<<<nblcks, nthrds>>>((double *) a,
				       (double *) b, *n);
  }
  
  /**
   * Fortran wrapper for col2
   * Vector multiplication with 2 vectors \f$ a = a \cdot b \f$
   */
  void cuda_col2(void *a, void *b, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    col2_kernel<<<nblcks, nthrds>>>((double *) a, 
				    (double *) b, *n);
  }
  
  /**
   * Fortran wrapper for col3
   * Vector multiplication with 3 vectors \f$ a = b \cdot c \f$
   */
  void cuda_col3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    col3_kernel<<<nblcks, nthrds>>>((double *) a, (double *) b,
				    (double *) c, *n);
  }
  

  /**
   * Fortran wrapper for sub3
   * Vector subtraction \f$ a = b - c \f$
   */
  void cuda_sub3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    sub3_kernel<<<nblcks, nthrds>>>((double *) a, (double *) b, 
				    (double *) c, *n);
  }

  /**
   * Fortran wrapper for addcol3
   * \f$ a = a + b * c \f$
   */
  void cuda_addcol3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    addcol3_kernel<<<nblcks, nthrds>>>((double *) a, (double *) b,
				       (double *) c, *n);
  }

  /**
   * Fortran wrapper glsc3
   * Weighted inner product \f$ a^T b c \f$
   */
  double cuda_glsc3(void *a, void *b, void *c, int *n) {
	
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const int nb = ((*n) + 1024 - 1)/ 1024;
    
    double * buf = (double *) malloc(nb * sizeof(double));
    double * buf_d;

    hipMalloc(&buf_d, nb*sizeof(double));
     
    glsc3_kernel<<<nblcks, nthrds>>>((double *) a, (double *) b,
				     (double *) c, buf_d, *n);

    hipMemcpy(buf, buf_d, nb * sizeof(double), hipMemcpyDeviceToHost);

    double res = 0.0;
    for (int i = 0; i < nb; i++) {
      res += buf[i];
    }

    free(buf);
    hipFree(buf_d);

    return res;
  }
}